#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/conv_layer.hpp"

namespace caffe {
template <typename Dtype>
__global__ void CCMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype thres0, Dtype thres1) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index]==1 && fabs(wb[index])<=thres0) 
      mask[index] = 0;
    else if (mask[index]==0 && fabs(wb[index])>thres1)
      mask[index] = 1;
  }
}

template <typename Dtype>
__global__ void CCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];    
  }
}
template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weightMask = this->blobs_[2]->mutable_gpu_data();
  Dtype* weightTmp = this->weight_tmp_.mutable_gpu_data(); 
  const Dtype* bias = NULL;
  Dtype* biasMask = NULL;
  Dtype* biasTmp = NULL; 
  Dtype thres0=0.0001,thres1=0.0005;
  if (this->bias_term_) {  
    bias = this->blobs_[1]->mutable_gpu_data();   
    biasMask = this->blobs_[3]->mutable_gpu_data();
    biasTmp = this->bias_tmp_.mutable_gpu_data();
  }
  if (this->phase_ == TRAIN){
	// Calculate the weight mask and bias mask with probability 
      CCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, 
        weightMask, thres0, thres1);
      CUDA_POST_KERNEL_CHECK;    
      if (this->bias_term_) {   
        CCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
          CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, 
          biasMask, thres0, thres1);
        CUDA_POST_KERNEL_CHECK; 
      }    
  }   

  // Calculate the current (masked) weight and bias
  CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, weightTmp);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {  
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, biasTmp);
    CUDA_POST_KERNEL_CHECK;  
  }
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weightTmp,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, biasTmp);
      }
    }
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const Dtype* weightTmp = this->weight_tmp_.gpu_data();  	
  const Dtype* weightMask = this->blobs_[2]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      const Dtype* biasMask = this->blobs_[3]->gpu_data();
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[3]->count(), bias_diff, biasMask, bias_diff);
      CUDA_POST_KERNEL_CHECK;  
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[2]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[2]->count(), weight_diff, weightMask, weight_diff);
      CUDA_POST_KERNEL_CHECK; 
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weightTmp,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
