#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void CCMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype thres0, Dtype thres1) {
  CUDA_KERNEL_LOOP(index, n) {
    if (mask[index]==1 && fabs(wb[index])<=thres0) 
      mask[index] = 0;
    else if (mask[index]==0 && fabs(wb[index])>thres1)
      mask[index] = 1;
  }
}

template <typename Dtype>
__global__ void CCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];    
  }
}

template <typename Dtype>
void InnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weightMask = this->masks_[0]->mutable_gpu_data();
  Dtype* weightTmp = this->weight_tmp_.mutable_gpu_data();  
  const Dtype* bias = NULL;
  Dtype* biasMask = NULL;
  Dtype* biasTmp = NULL;
  Dtype thres0,thres1;
  if (this->bias_term_) {  
    bias = this->blobs_[1]->mutable_gpu_data();   
    biasMask = this->masks_[1]->mutable_gpu_data();
    biasTmp = this->bias_tmp_.mutable_gpu_data();
  }   
  if (this->phase_ == TRAIN){
      CCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, thres0,thres1);
      CUDA_POST_KERNEL_CHECK;    
      if (this->bias_term_) {  
        CCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
          CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, thres0,thres1);
        CUDA_POST_KERNEL_CHECK;  
      }    
  }  
  
  // Calculate the current (masked) weight and bias
  CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, weightTmp);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {  
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, biasTmp);
    CUDA_POST_KERNEL_CHECK;  
  } 
  
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weightTmp, bottom_data, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            biasTmp, top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          transpose_ ? CblasNoTrans : CblasTrans,
                          M_, N_, K_, (Dtype)1.,
                          bottom_data, weightTmp, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            biasTmp, (Dtype)1., top_data);
  }
}

template <typename Dtype>
void InnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* weightMask = this->masks_[0]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    // Gradient with respect to weight
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->masks_[0]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->masks_[0]->count(), weight_diff, weightMask, weight_diff);
    CUDA_POST_KERNEL_CHECK; 
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          K_, N_, M_,
          (Dtype)1., bottom_data, top_diff,
          (Dtype)1., weight_diff);
    } else {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff, bottom_data,
          (Dtype)1., weight_diff);
    }
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* biasMask = this->masks_[1]->gpu_data();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    // Gradient with respect to bias
    CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->masks_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->masks_[1]->count(), bias_diff, biasMask, bias_diff);
    CUDA_POST_KERNEL_CHECK;
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,bias_diff);
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* weightTmp = this->weight_tmp_.gpu_data();
    // Gradient with respect to bottom data
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
          M_, K_, N_,
          (Dtype)1., top_diff, weightTmp,
          (Dtype)0., bottom[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
          M_, K_, N_,
         (Dtype)1., top_diff, weightTmp,
         (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductLayer);

}  // namespace caffe
